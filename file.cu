
#include <hip/hip_runtime.h>
#include <string>
#include <vector>
#include <iostream>

#include <algorithm>
#include <dirent.h>
#include "sys/stat.h"

using namespace std;

void scanDir(const string& base_path, const string& dir_path, std::vector<std::string>& filenames) {
    DIR *pDir;
    struct dirent* ptr;
    if(!(pDir = opendir(std::string(base_path).append(dir_path).c_str()))){
        std::cout<<"Folder doesn't Exist!"<<std::endl;
        return;
    }
    struct stat s_buff;

    while((ptr = readdir(pDir)) != nullptr) {
        std::string fileName(ptr->d_name);
        std::string abs_path = std::string(base_path).append("/").append(dir_path).append("/").append(fileName);
        std::string relative_path = std::string(dir_path).append("/").append(fileName);

        std::string extension = fileName.substr(fileName.find_last_of(".") + 1);
        transform(extension.begin(), extension.end(), extension.begin(), ::tolower);

        // ignore temp files
        if (extension.compare("tmp") == 0) {
            continue;
        }

        if (strcmp(ptr->d_name, ".") == 0 || strcmp(ptr->d_name, "..") == 0){
            continue;
        }

        stat(abs_path.c_str(), &s_buff);
        if (S_ISREG(s_buff.st_mode)) {
            filenames.push_back(relative_path);
            continue;
        }

        if (S_ISDIR(s_buff.st_mode)) {
            scanDir(base_path, relative_path, filenames);
            continue;
        }
    }
    closedir(pDir);
}

void readDir(const char* dirPath, std::vector<std::string>& filenames) {

    scanDir(std::string(dirPath), std::string(""), filenames);

//    DIR *pDir;
//    struct dirent* ptr;
//    if(!(pDir = opendir(dirPath))){
//        std::cout<<"Folder doesn't Exist!"<<std::endl;
//        return;
//    }
//    struct stat s_buff;
//
//    while((ptr = readdir(pDir)) != nullptr) {
//        std::string fileName(ptr->d_name);
//        std::string path = std::string(dirPath) + "/" + fileName;
//
//        std::string extension = fileName.substr(fileName.find_last_of(".") + 1);
//        transform(extension.begin(), extension.end(), extension.begin(), ::tolower);
//        printf("%s", ptr->d_name);
//        // ignore temp files
//        if (!extension.compare("tmp") == 0) {
//            continue;
//        }
//
//        if (strcmp(ptr->d_name, ".") == 0 || strcmp(ptr->d_name, "..") == 0){
//            continue;
//        }
//
//        stat(path.c_str(), &s_buff);
//        if (!S_ISREG(s_buff.st_mode)) {
//            continue;
//        }
//
//        filenames.push_back(fileName);
//    }
//    closedir(pDir);
}