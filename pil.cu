#include "hip/hip_runtime.h"
//
// Created by siderzhangPC on 2024/4/19.
//

#include "ipl.h"
#include "Imaging.h"
#include <math.h>
#include <cstdio>


#define MAKE_UINT32(u0, u1, u2, u3) \
    ((UINT32)(u0) | ((UINT32)(u1) << 8) | ((UINT32)(u2) << 16) | ((UINT32)(u3) << 24))

/* Handles values form -640 to 639. */
UINT8 _clip8_lookups[1280] = {
        0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
        0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
        0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
        0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
        0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
        0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
        0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
        0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
        0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
        0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
        0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
        0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
        0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
        0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
        0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
        0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
        0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
        0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
        0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
        0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
        0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
        0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
        0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
        0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
        0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
        0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
        0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
        0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
        0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
        0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
        0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
        0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
        0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
        0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
        0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
        0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
        0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
        0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   1,   2,   3,   4,   5,
        6,   7,   8,   9,   10,  11,  12,  13,  14,  15,  16,  17,  18,  19,  20,  21,  22,
        23,  24,  25,  26,  27,  28,  29,  30,  31,  32,  33,  34,  35,  36,  37,  38,  39,
        40,  41,  42,  43,  44,  45,  46,  47,  48,  49,  50,  51,  52,  53,  54,  55,  56,
        57,  58,  59,  60,  61,  62,  63,  64,  65,  66,  67,  68,  69,  70,  71,  72,  73,
        74,  75,  76,  77,  78,  79,  80,  81,  82,  83,  84,  85,  86,  87,  88,  89,  90,
        91,  92,  93,  94,  95,  96,  97,  98,  99,  100, 101, 102, 103, 104, 105, 106, 107,
        108, 109, 110, 111, 112, 113, 114, 115, 116, 117, 118, 119, 120, 121, 122, 123, 124,
        125, 126, 127, 128, 129, 130, 131, 132, 133, 134, 135, 136, 137, 138, 139, 140, 141,
        142, 143, 144, 145, 146, 147, 148, 149, 150, 151, 152, 153, 154, 155, 156, 157, 158,
        159, 160, 161, 162, 163, 164, 165, 166, 167, 168, 169, 170, 171, 172, 173, 174, 175,
        176, 177, 178, 179, 180, 181, 182, 183, 184, 185, 186, 187, 188, 189, 190, 191, 192,
        193, 194, 195, 196, 197, 198, 199, 200, 201, 202, 203, 204, 205, 206, 207, 208, 209,
        210, 211, 212, 213, 214, 215, 216, 217, 218, 219, 220, 221, 222, 223, 224, 225, 226,
        227, 228, 229, 230, 231, 232, 233, 234, 235, 236, 237, 238, 239, 240, 241, 242, 243,
        244, 245, 246, 247, 248, 249, 250, 251, 252, 253, 254, 255, 255, 255, 255, 255, 255,
        255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
        255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
        255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
        255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
        255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
        255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
        255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
        255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
        255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
        255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
        255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
        255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
        255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
        255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
        255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
        255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
        255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
        255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
        255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
        255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
        255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
        255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
        255, 255, 255, 255, 255,
};

UINT8 *clip8_lookups = &_clip8_lookups[640];

static inline UINT8
clip8(int in) {
    return clip8_lookups[in >> PRECISION_BITS];
}

static inline double
bilinear_filter(double x) {
    if (x < 0.0) {
        x = -x;
    }
    if (x < 1.0) {
        return 1.0 - x;
    }
    return 0.0;
}

static struct filter BILINEAR = {bilinear_filter, 1.0};

int
precompute_coeffs(
        int inSize,
        float in0,
        float in1,
        int outSize,
//        struct filter *filterp,
        int **boundsp,
        double **kkp) {
    double support, scale, filterscale;
    double center, ww, ss;
    int xx, x, ksize, xmin, xmax;
    int *bounds;
    double *kk, *k;

    // panding
    struct filter *filterp = &BILINEAR;

    /* prepare for horizontal stretch */
    filterscale = scale = (double)(in1 - in0) / outSize;
    if (filterscale < 1.0) {
        filterscale = 1.0;
    }

    /* determine support size (length of resampling filter) */
    support = filterp->support * filterscale;

    /* maximum number of coeffs */
    ksize = (int)ceil(support) * 2 + 1;

    // check for overflow
    if (outSize > INT_MAX / (ksize * (int)sizeof(double))) {
        return 0;
    }

    /* coefficient buffer */
    /* malloc check ok, overflow checked above */
    kk = (double*)malloc(outSize * ksize * sizeof(double));
    if (!kk) {
        return 0;
    }

    /* malloc check ok, ksize*sizeof(double) > 2*sizeof(int) */
    bounds = (int*)malloc(outSize * 2 * sizeof(int));
    if (!bounds) {
        free(kk);
        return 0;
    }

    for (xx = 0; xx < outSize; xx++) {
        center = in0 + (xx + 0.5) * scale;
        ww = 0.0;
        ss = 1.0 / filterscale;
        // Round the value
        xmin = (int)(center - support + 0.5);
        if (xmin < 0) {
            xmin = 0;
        }
        // Round the value
        xmax = (int)(center + support + 0.5);
        if (xmax > inSize) {
            xmax = inSize;
        }
        xmax -= xmin;
        k = &kk[xx * ksize];
        for (x = 0; x < xmax; x++) {
            double w = filterp->filter((x + xmin - center + 0.5) * ss);
            k[x] = w;
            ww += w;
        }
        for (x = 0; x < xmax; x++) {
            if (ww != 0.0) {
                k[x] /= ww;
            }
        }
        // Remaining values should stay empty if they are used despite of xmax.
        for (; x < ksize; x++) {
            k[x] = 0;
        }
        bounds[xx * 2 + 0] = xmin;
        bounds[xx * 2 + 1] = xmax;
    }
    *boundsp = bounds;
    *kkp = kk;
    return ksize;
}


void
normalize_coeffs_8bpc(int outSize, int ksize, double *prekk) {
    int x;
    INT32 *kk;

    // use the same buffer for normalized coefficients
    kk = (INT32 *)prekk;

    for (x = 0; x < outSize * ksize; x++) {

        if (x == 15) {
            int sd = 0;
        }
        if (prekk[x] < 0) {
            kk[x] = (int)(-0.5 + prekk[x] * (1 << PRECISION_BITS));
        } else {
            kk[x] = (int)(0.5 + prekk[x] * (1 << PRECISION_BITS));
        }
    }
}

void
ImagingResampleHorizontal_8bpc(
        Imaging imOut, Imaging imIn, int offset, int ksize, int *bounds, double *prekk) {
//    ImagingSectionCookie cookie;
    int ss0, ss1, ss2, ss3;
    int xx, yy, x, xmin, xmax;
    INT32 *k, *kk;

    // use the same buffer for normalized coefficients
    kk = (INT32 *)prekk;
    normalize_coeffs_8bpc(imOut->xsize, ksize, prekk);

//    ImagingSectionEnter(&cookie);
    if (imIn->image8) {
        for (yy = 0; yy < imOut->ysize; yy++) {
            for (xx = 0; xx < imOut->xsize; xx++) {
                xmin = bounds[xx * 2 + 0];
                xmax = bounds[xx * 2 + 1];
                k = &kk[xx * ksize];
                ss0 = 1 << (PRECISION_BITS - 1);
                for (x = 0; x < xmax; x++) {
                    ss0 += ((UINT8)imIn->image8[yy + offset][x + xmin]) * k[x];
                }
                imOut->image8[yy][xx] = clip8(ss0);
            }
        }
    } else if (imIn->type == IMAGING_TYPE_UINT8) {
        if (imIn->bands == 2) {
            for (yy = 0; yy < imOut->ysize; yy++) {
                for (xx = 0; xx < imOut->xsize; xx++) {
                    UINT32 v;
                    xmin = bounds[xx * 2 + 0];
                    xmax = bounds[xx * 2 + 1];
                    k = &kk[xx * ksize];
                    ss0 = ss3 = 1 << (PRECISION_BITS - 1);
                    for (x = 0; x < xmax; x++) {
                        ss0 += ((UINT8)imIn->image[yy + offset][(x + xmin) * 4 + 0]) *
                               k[x];
                        ss3 += ((UINT8)imIn->image[yy + offset][(x + xmin) * 4 + 3]) *
                               k[x];
                    }
                    v = MAKE_UINT32(clip8(ss0), 0, 0, clip8(ss3));
                    memcpy(imOut->image[yy] + xx * sizeof(v), &v, sizeof(v));
                }
            }
        } else if (imIn->bands == 3) {
            for (yy = 0; yy < imOut->ysize; yy++) {
                for (xx = 0; xx < imOut->xsize; xx++) {
                    UINT32 v;
                    xmin = bounds[xx * 2 + 0];
                    xmax = bounds[xx * 2 + 1];
                    k = &kk[xx * ksize];
                    ss0 = ss1 = ss2 = 1 << (PRECISION_BITS - 1);
                    for (x = 0; x < xmax; x++) {
                        ss0 += ((UINT8)imIn->image[yy + offset][(x + xmin) * 4 + 0]) *
                               k[x];
                        ss1 += ((UINT8)imIn->image[yy + offset][(x + xmin) * 4 + 1]) *
                               k[x];
                        ss2 += ((UINT8)imIn->image[yy + offset][(x + xmin) * 4 + 2]) *
                               k[x];
                    }
                    UINT8 as = clip8(ss0);
                    v = MAKE_UINT32(clip8(ss0), clip8(ss1), clip8(ss2), 0);
                    memcpy(imOut->image[yy] + xx * sizeof(v), &v, sizeof(v));
                }
            }
        } else {
            for (yy = 0; yy < imOut->ysize; yy++) {
                for (xx = 0; xx < imOut->xsize; xx++) {
                    UINT32 v;
                    xmin = bounds[xx * 2 + 0];
                    xmax = bounds[xx * 2 + 1];
                    k = &kk[xx * ksize];
                    ss0 = ss1 = ss2 = ss3 = 1 << (PRECISION_BITS - 1);
                    for (x = 0; x < xmax; x++) {
                        ss0 += ((UINT8)imIn->image[yy + offset][(x + xmin) * 4 + 0]) *
                               k[x];
                        ss1 += ((UINT8)imIn->image[yy + offset][(x + xmin) * 4 + 1]) *
                               k[x];
                        ss2 += ((UINT8)imIn->image[yy + offset][(x + xmin) * 4 + 2]) *
                               k[x];
                        ss3 += ((UINT8)imIn->image[yy + offset][(x + xmin) * 4 + 3]) *
                               k[x];
                    }
                    v = MAKE_UINT32(clip8(ss0), clip8(ss1), clip8(ss2), clip8(ss3));
                    memcpy(imOut->image[yy] + xx * sizeof(v), &v, sizeof(v));
                }
            }
        }
    }
//    ImagingSectionLeave(&cookie);
}

void
ImagingResampleVertical_8bpc(
        Imaging imOut, Imaging imIn, int offset, int ksize, int *bounds, double *prekk) {
//    ImagingSectionCookie cookie;
    int ss0, ss1, ss2, ss3;
    int xx, yy, y, ymin, ymax;
    INT32 *k, *kk;

    // use the same buffer for normalized coefficients
    kk = (INT32 *)prekk;
    normalize_coeffs_8bpc(imOut->ysize, ksize, prekk);

//    ImagingSectionEnter(&cookie);
    if (imIn->image8) {
        for (yy = 0; yy < imOut->ysize; yy++) {
            k = &kk[yy * ksize];
            ymin = bounds[yy * 2 + 0];
            ymax = bounds[yy * 2 + 1];
            for (xx = 0; xx < imOut->xsize; xx++) {
                ss0 = 1 << (PRECISION_BITS - 1);
                for (y = 0; y < ymax; y++) {
                    ss0 += ((UINT8)imIn->image8[y + ymin][xx]) * k[y];
                }
                imOut->image8[yy][xx] = clip8(ss0);
            }
        }
    } else if (imIn->type == IMAGING_TYPE_UINT8) {
        if (imIn->bands == 2) {
            for (yy = 0; yy < imOut->ysize; yy++) {
                k = &kk[yy * ksize];
                ymin = bounds[yy * 2 + 0];
                ymax = bounds[yy * 2 + 1];
                for (xx = 0; xx < imOut->xsize; xx++) {
                    UINT32 v;
                    ss0 = ss3 = 1 << (PRECISION_BITS - 1);
                    for (y = 0; y < ymax; y++) {
                        ss0 += ((UINT8)imIn->image[y + ymin][xx * 4 + 0]) * k[y];
                        ss3 += ((UINT8)imIn->image[y + ymin][xx * 4 + 3]) * k[y];
                    }
                    v = MAKE_UINT32(clip8(ss0), 0, 0, clip8(ss3));
                    memcpy(imOut->image[yy] + xx * sizeof(v), &v, sizeof(v));
                }
            }
        } else if (imIn->bands == 3) {
            for (yy = 0; yy < imOut->ysize; yy++) {
                k = &kk[yy * ksize];
                ymin = bounds[yy * 2 + 0];
                ymax = bounds[yy * 2 + 1];
                for (xx = 0; xx < imOut->xsize; xx++) {
                    UINT32 v;
                    ss0 = ss1 = ss2 = 1 << (PRECISION_BITS - 1);
                    for (y = 0; y < ymax; y++) {
                        ss0 += ((UINT8)imIn->image[y + ymin][xx * 4 + 0]) * k[y];
                        ss1 += ((UINT8)imIn->image[y + ymin][xx * 4 + 1]) * k[y];
                        ss2 += ((UINT8)imIn->image[y + ymin][xx * 4 + 2]) * k[y];
                    }
                    v = MAKE_UINT32(clip8(ss0), clip8(ss1), clip8(ss2), 0);
                    memcpy(imOut->image[yy] + xx * sizeof(v), &v, sizeof(v));
                }
            }
        } else {
            for (yy = 0; yy < imOut->ysize; yy++) {
                k = &kk[yy * ksize];
                ymin = bounds[yy * 2 + 0];
                ymax = bounds[yy * 2 + 1];
                for (xx = 0; xx < imOut->xsize; xx++) {
                    UINT32 v;
                    ss0 = ss1 = ss2 = ss3 = 1 << (PRECISION_BITS - 1);
                    for (y = 0; y < ymax; y++) {
                        ss0 += ((UINT8)imIn->image[y + ymin][xx * 4 + 0]) * k[y];
                        ss1 += ((UINT8)imIn->image[y + ymin][xx * 4 + 1]) * k[y];
                        ss2 += ((UINT8)imIn->image[y + ymin][xx * 4 + 2]) * k[y];
                        ss3 += ((UINT8)imIn->image[y + ymin][xx * 4 + 3]) * k[y];
                    }
                    v = MAKE_UINT32(clip8(ss0), clip8(ss1), clip8(ss2), clip8(ss3));
                    memcpy(imOut->image[yy] + xx * sizeof(v), &v, sizeof(v));
                }
            }
        }
    }
//    ImagingSectionLeave(&cookie);
}

Imaging getImage(int width, int height, unsigned char* data) {
    Imaging imaging = (Imaging)calloc(1, sizeof(ImagingMemoryInstance));
    memcpy(imaging->mode, "RGB", sizeof("RGB"));
    imaging->bands = 3;
    imaging->xsize = width;
    imaging->ysize = height;
    imaging->image = static_cast<char **>(calloc(imaging->ysize, sizeof(char *)));
    for (int i = 0;i < imaging->ysize;i++) {
        imaging->image[i] = (char*)calloc(imaging->xsize * 4, sizeof(UINT8));
        if (data != NULL) {
            for (int j = 0;j < imaging->xsize;j++) {
                imaging->image[i][j * 4 + 0] = data[i * imaging->xsize * 3 + j * 3 + 0];
                imaging->image[i][j * 4 + 1] = data[i * imaging->xsize * 3 + j * 3 + 1];
                imaging->image[i][j * 4 + 2] = data[i * imaging->xsize * 3 + j * 3 + 2];
                imaging->image[i][j * 4 + 3] = 0;
            }
        }
    }

    return imaging;
}