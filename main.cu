#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <stdio.h>
#include "jpeg.h"
#include <vector>
#include <string.h>
#include <dirent.h>
#include "npy.h"
#include "sys/types.h"
#include "sys/stat.h"
#include <algorithm>

#include <sys/timeb.h>
#include <unistd.h>

#define PRECISION_BITS (32 - 8 - 2)


typedef signed char         INT8, *PINT8;
typedef signed short        INT16, *PINT16;
typedef signed int          INT32, *PINT32;
typedef unsigned char       UINT8, *PUINT8;
typedef unsigned short      UINT16, *PUINT16;
typedef unsigned int        UINT32, *PUINT32;

__global__ void coeffs(int inSize, int in0, int in1,
                       int outSize, int *boundsp, double *kkp) {
    double scale, filterscale;
    double center, ww, ss;
    int x, ksize, xmin, xmax;
    int *bounds;
    double *kk, *k;

    /* prepare for horizontal stretch */
    filterscale = scale = (double)(in1 - in0) / outSize;
    if (filterscale < 1.0) {
        filterscale = 1.0;
    }

    /* determine support size (length of resampling filter) */
    double support = 1.0 * filterscale;

    /* maximum number of coeffs */
    ksize = (int)ceil(support) * 2 + 1;

    // check for overflow
    if (outSize > INT_MAX / (ksize * (int)sizeof(double))) {
        return;
    }

    /* coefficient buffer */
    /* malloc check ok, overflow checked above */
    kk = kkp;

    /* malloc check ok, ksize*sizeof(double) > 2*sizeof(int) */
    bounds = boundsp;
    int _xx = blockIdx.x * blockDim.x + threadIdx.x;
    if (_xx >= outSize) {
        return;
    }

    for (unsigned int xx = _xx;xx < outSize;xx += gridDim.x * blockDim.x) {
        center = in0 + (xx + 0.5) * scale;
        ww = 0.0;
        ss = 1.0 / filterscale;
        // Round the value
        xmin = (int) (center - support + 0.5);
        if (xmin < 0) {
            xmin = 0;
        }
        // Round the value
        xmax = (int) (center + support + 0.5);
        if (xmax > inSize) {
            xmax = inSize;
        }
        xmax -= xmin;
        k = &kk[xx * ksize];
        for (x = 0; x < xmax; x++) {
            double filteringX = (x + xmin - center + 0.5) * ss;

            double w = 0;//filterp->filter((x + xmin - center + 0.5) * ss);

            if (filteringX < 0.0) {
                filteringX = -filteringX;
            }
            if (filteringX < 1.0) {
                w = 1.0 - filteringX;
            } else {
                w = 0.0;
            }

            k[x] = w;
            ww += w;
        }
        for (x = 0; x < xmax; x++) {
            if (ww != 0.0) {
                k[x] /= ww;
            }
        }
        // Remaining values should stay empty if they are used despite of xmax.
        for (; x < ksize; x++) {
            k[x] = 0;
        }

        bounds[xx * 2 + 0] = xmin;
        bounds[xx * 2 + 1] = xmax;
    }
}


__global__ void normalize_coeffs(int outSize, int ksize, double *prekk) {
    INT32 *kk;

    // use the same buffer for normalized coefficients
    kk = (INT32 *)prekk;

    unsigned int _x = blockIdx.x * blockDim.x + threadIdx.x;
    if (_x >= outSize * ksize) {
        return;
    }


    for (unsigned int x = _x;x < outSize * ksize;x += gridDim.x * blockDim.x) {
        if (prekk[x] < 0) {
            kk[x] = (int) (-0.5 + prekk[x] * (1 << PRECISION_BITS));
        } else {
            kk[x] = (int) (0.5 + prekk[x] * (1 << PRECISION_BITS));
        }
    }
}

__global__ void shift_ysize(int *boundsp, int ysize) {
    unsigned int _i = blockIdx.x * blockDim.x + threadIdx.x;
    if (_i >= ysize) {
        return;
    }
//    printf("Bound %d %d\n", boundsp[i * 2], boundsp[0]);

    // Shift bounds for vertical pass
    for (unsigned int i = _i;i < ysize;i += gridDim.x * blockDim.x) {
        boundsp[i * 2] -= boundsp[0];
    }
}

__global__ void build_result_horiz(int srcXsize, int channelCount, int xsize, int ysize,
                                   unsigned char* input, unsigned char* output, int ksize, int *bounds, double *prekk, unsigned char* _lookups) {
    unsigned int _id = blockIdx.x * blockDim.x + threadIdx.x;
    if (_id >= xsize * ysize) {
        return;
    }

    for (unsigned int id = _id;id < xsize * ysize;id += gridDim.x * blockDim.x) {

        unsigned int xx = id % xsize;
        unsigned int yy = id / xsize;

        int ss0, ss1, ss2;
        INT32 *kk = (INT32 *) prekk;

        int xmin = bounds[xx * 2 + 0];
        int xmax = bounds[xx * 2 + 1];
        INT32 *k = &kk[xx * ksize];
        ss0 = ss1 = ss2 = 1 << (PRECISION_BITS - 1);

        for (int x = 0; x < xmax; x++) {

            ss0 += ((UINT8) input[yy * srcXsize * channelCount + (x + xmin) * channelCount + 0]) *
                   k[x];
            ss1 += ((UINT8) input[yy * srcXsize * channelCount + (x + xmin) * channelCount + 1]) *
                   k[x];
            ss2 += ((UINT8) input[yy * srcXsize * channelCount + (x + xmin) * channelCount + 2]) *
                   k[x];
        }

        UINT8 *lookups = &_lookups[640];
        UINT8 ss0_1 = lookups[ss0 >> PRECISION_BITS];
        UINT8 ss1_1 = lookups[ss1 >> PRECISION_BITS];
        UINT8 ss2_1 = lookups[ss2 >> PRECISION_BITS];

        output[yy * xsize * channelCount + xx * channelCount + 0] = ss0_1;
        output[yy * xsize * channelCount + xx * channelCount + 1] = ss1_1;
        output[yy * xsize * channelCount + xx * channelCount + 2] = ss2_1;
    }
}

__global__ void build_result_vert(int srcXsize, int channelCount, int xsize, int ysize,
                                  unsigned char* input, unsigned char* output, int ksize, int *bounds, double *prekk, unsigned char* _lookups) {
    unsigned int _id = blockIdx.x * blockDim.x + threadIdx.x;
    if (_id >= xsize * ysize) {
        return;
    }

    for (unsigned int id = _id;id < xsize * ysize;id += gridDim.x * blockDim.x) {
        unsigned int xx = id % xsize;
        unsigned int yy = id / xsize;

        int ss0, ss1, ss2;
        INT32 *kk = (INT32 *) prekk;

        INT32 *k = &kk[yy * ksize];
        int ymin = bounds[yy * 2 + 0];
        int ymax = bounds[yy * 2 + 1];

        ss0 = ss1 = ss2 = 1 << (PRECISION_BITS - 1);
        for (int y = 0; y < ymax; y++) {
            ss0 += ((UINT8) input[(y + ymin) * srcXsize * channelCount + xx * channelCount + 0]) * k[y];
            ss1 += ((UINT8) input[(y + ymin) * srcXsize * channelCount + xx * channelCount + 1]) * k[y];
            ss2 += ((UINT8) input[(y + ymin) * srcXsize * channelCount + xx * channelCount + 2]) * k[y];
        }

        UINT8 *lookups = &_lookups[640];
        UINT8 ss0_1 = lookups[ss0 >> PRECISION_BITS];
        UINT8 ss1_1 = lookups[ss1 >> PRECISION_BITS];
        UINT8 ss2_1 = lookups[ss2 >> PRECISION_BITS];
        output[yy * xsize * channelCount + xx * channelCount + 0] = ss0_1;
        output[yy * xsize * channelCount + xx * channelCount + 1] = ss1_1;
        output[yy * xsize * channelCount + xx * channelCount + 2] = ss2_1;
    }
}

int *bounds_horiz_d;
double *kk_horiz_d;
int ksize_horiz;

int  *bounds_vert_d;
double *kk_vert_d;
int ksize_vert;

void coffes(unsigned int im_xsize, unsigned int im_ysize, int xsize, int ysize) {

    float box[4] = {0, 0, 1.0f * im_xsize, 1.0f * im_ysize};

    double filterscale_horiz = (double)(box[2] - box[0]) / xsize;
    if (filterscale_horiz < 1.0) {
        filterscale_horiz = 1.0;
    }
    ksize_horiz = (int)ceil(filterscale_horiz) * 2 + 1;

    hipMalloc(&kk_horiz_d, xsize * ksize_horiz * sizeof(double));
    hipMalloc(&bounds_horiz_d, xsize * 2 * sizeof(int));

    coeffs<<<256, 256>>>(im_xsize,
                        box[0],
                        box[2],
                        xsize,
                        bounds_horiz_d,
                        kk_horiz_d);

    normalize_coeffs<<<xsize, ksize_horiz>>>(xsize, ksize_horiz, kk_horiz_d);

    double filterscale_vert = (double)(box[3] - box[1]) / ysize;
    if (filterscale_vert < 1.0) {
        filterscale_vert = 1.0;
    }
    ksize_vert = (int)ceil(filterscale_vert) * 2 + 1;
    hipMalloc(&kk_vert_d, ysize * ksize_vert * sizeof(double));
    hipMalloc(&bounds_vert_d, ysize * 2 * sizeof(int));

    coeffs<<<256, 256>>>(im_ysize,
                        box[1],
                        box[3],
                        ysize,
                        bounds_vert_d,
                        kk_vert_d);

    normalize_coeffs<<<ysize, ksize_vert>>>(ysize, ksize_vert, kk_vert_d);

    shift_ysize<<<256, 256>>>(bounds_vert_d, ysize);
}

__global__ void rescale_normalize_d(unsigned char *data, float* result, int xsize, int ysize) {
    unsigned int _id = blockIdx.x * blockDim.x + threadIdx.x;

    if (_id >= xsize * ysize) {
        return;
    }

    for (unsigned int id = _id;id < xsize * ysize;id += gridDim.x * blockDim.x) {
        // rescale
        double pixel0 = ((double)data[id * 3 + 0]) * 1.0 / 255;
        double pixel1 = ((double)data[id * 3 + 1]) * 1.0 / 255;
        double pixel2 = ((double)data[id * 3 + 2]) * 1.0 / 255;

        // normalize
        pixel0 = (pixel0 - 0.5) / 0.5;
        pixel1 = (pixel1 - 0.5) / 0.5;
        pixel2 = (pixel2 - 0.5) / 0.5;

        // transpose
        result[id + 0 * xsize * ysize] = (float)pixel0;
        result[id + 1 * xsize * ysize] = (float)pixel1;
        result[id + 2 * xsize * ysize] = (float)pixel2;
    }
}

UINT8 lookups_h[1280] = {
        0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
        0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
        0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
        0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
        0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
        0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
        0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
        0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
        0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
        0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
        0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
        0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
        0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
        0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
        0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
        0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
        0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
        0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
        0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
        0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
        0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
        0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
        0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
        0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
        0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
        0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
        0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
        0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
        0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
        0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
        0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
        0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
        0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
        0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
        0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
        0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
        0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
        0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   1,   2,   3,   4,   5,
        6,   7,   8,   9,   10,  11,  12,  13,  14,  15,  16,  17,  18,  19,  20,  21,  22,
        23,  24,  25,  26,  27,  28,  29,  30,  31,  32,  33,  34,  35,  36,  37,  38,  39,
        40,  41,  42,  43,  44,  45,  46,  47,  48,  49,  50,  51,  52,  53,  54,  55,  56,
        57,  58,  59,  60,  61,  62,  63,  64,  65,  66,  67,  68,  69,  70,  71,  72,  73,
        74,  75,  76,  77,  78,  79,  80,  81,  82,  83,  84,  85,  86,  87,  88,  89,  90,
        91,  92,  93,  94,  95,  96,  97,  98,  99,  100, 101, 102, 103, 104, 105, 106, 107,
        108, 109, 110, 111, 112, 113, 114, 115, 116, 117, 118, 119, 120, 121, 122, 123, 124,
        125, 126, 127, 128, 129, 130, 131, 132, 133, 134, 135, 136, 137, 138, 139, 140, 141,
        142, 143, 144, 145, 146, 147, 148, 149, 150, 151, 152, 153, 154, 155, 156, 157, 158,
        159, 160, 161, 162, 163, 164, 165, 166, 167, 168, 169, 170, 171, 172, 173, 174, 175,
        176, 177, 178, 179, 180, 181, 182, 183, 184, 185, 186, 187, 188, 189, 190, 191, 192,
        193, 194, 195, 196, 197, 198, 199, 200, 201, 202, 203, 204, 205, 206, 207, 208, 209,
        210, 211, 212, 213, 214, 215, 216, 217, 218, 219, 220, 221, 222, 223, 224, 225, 226,
        227, 228, 229, 230, 231, 232, 233, 234, 235, 236, 237, 238, 239, 240, 241, 242, 243,
        244, 245, 246, 247, 248, 249, 250, 251, 252, 253, 254, 255, 255, 255, 255, 255, 255,
        255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
        255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
        255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
        255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
        255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
        255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
        255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
        255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
        255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
        255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
        255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
        255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
        255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
        255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
        255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
        255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
        255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
        255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
        255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
        255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
        255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
        255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
        255, 255, 255, 255, 255,
};

extern "C" int resize(const char* filename, std::string& outputFilename, unsigned int target_xsize, unsigned int target_ysize) {
    unsigned char *image_pixel_data = NULL;
    unsigned int source_xsize;
    unsigned int source_ysize;
    unsigned int channels;

    int ret = read_jpeg_file(filename, &image_pixel_data, &source_xsize, &source_ysize, &channels);
    if (ret != 0) {
        std::cerr<<"failed to load file" << filename <<std::endl;
        return -1;
    }

    coffes(source_xsize, source_ysize, target_xsize, target_ysize);

    unsigned char* input;
    hipMalloc(&input, source_xsize * source_ysize * channels * sizeof(unsigned char));

    hipMemcpy(input, image_pixel_data, source_xsize * source_ysize * channels * sizeof(unsigned char), hipMemcpyHostToDevice);

    unsigned char* temp;
    hipMalloc(&temp, target_xsize * source_ysize * channels * sizeof(unsigned char));

    unsigned char* result_pixel_data_d;
    hipMalloc(&result_pixel_data_d, target_xsize * target_ysize * channels * sizeof(unsigned char));

    unsigned char* looksup_d;
    hipMalloc(&looksup_d, 1280 * sizeof(unsigned char));
    hipMemcpy(looksup_d, lookups_h, 1280 * sizeof(unsigned char), hipMemcpyHostToDevice);

    build_result_horiz<<<256, 1024>>>(source_xsize, channels, target_xsize, source_ysize, input, temp, ksize_horiz, bounds_horiz_d, kk_horiz_d, looksup_d);
    build_result_vert<<<256, 1024>>>(target_xsize, channels, target_xsize, target_ysize, temp, result_pixel_data_d, ksize_vert, bounds_vert_d, kk_vert_d, looksup_d);

    float* normalized_pixel_data_d;
    hipMalloc(&normalized_pixel_data_d, target_xsize * target_ysize * channels * sizeof(float));

    rescale_normalize_d<<<256, 1024>>>(result_pixel_data_d, normalized_pixel_data_d, target_xsize, target_ysize);

    std::vector<float> v;
    v.resize(target_xsize * target_ysize * 3);
    float* normalized_pixel_data = (float*) calloc(target_xsize * target_ysize, channels * sizeof(float));
    hipMemcpy(&v[0], normalized_pixel_data_d, target_xsize * target_ysize * channels * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(bounds_horiz_d);
    hipFree(kk_horiz_d);
    hipFree(bounds_vert_d);
    hipFree(kk_vert_d);

    hipFree(input);
    hipFree(temp);
    hipFree(looksup_d);
    hipFree(result_pixel_data_d);
    hipFree(normalized_pixel_data_d);

    hipDeviceSynchronize();


    const std::vector<unsigned long> leshape11{3, target_xsize, target_ysize};

    const npy::npy_data<float> data11{v, leshape11, false};
    std::string tmpFileName = outputFilename + ".tmp";
    write_npy(tmpFileName, data11);
    rename(tmpFileName.c_str(), outputFilename.c_str());

    return 0;
}

void readDir(const char* dirPath, std::vector<std::string>& filenames) {
    DIR *pDir;
    struct dirent* ptr;
    if(!(pDir = opendir(dirPath))){
        std::cout<<"Folder doesn't Exist!"<<std::endl;
        return;
    }
    struct stat s_buff;

    while((ptr = readdir(pDir)) != nullptr) {
        std::string fileName(ptr->d_name);
        std::string path = std::string(dirPath) + "/" + fileName;

        std::string extension = fileName.substr(fileName.find_last_of(".") + 1);
        transform(extension.begin(), extension.end(), extension.begin(), ::tolower);

        if (!extension.compare("jpg") == 0 && !extension.compare("jpeg") == 0) {
            continue;
        }

        if (strcmp(ptr->d_name, ".") == 0 || strcmp(ptr->d_name, "..") == 0){
            continue;
        }

        stat(path.c_str(), &s_buff);
        if (!S_ISREG(s_buff.st_mode)) {
            continue;
        }

        filenames.push_back(fileName);
    }
    closedir(pDir);
}

int main(int argc, char *argv[]) {
    if (argc < 4) {
        std::cerr<<"arguments: dirPath width height outputfilename"<<std::endl;
        return -1;
    }
    char* dirPath = argv[1];
    char* outputFilename = argv[2];
    unsigned int xsize = std::atoi(argv[3]);
    unsigned int ysize = std::atoi(argv[4]);

    while (true) {
        std::vector<std::string> fileNames;
        readDir(dirPath, fileNames);

        timeb t;
        ftime(&t);
        long t1 = t.time * 1000 + t.millitm;
        int count = 0;
        for (auto iter = fileNames.begin();iter != fileNames.end();iter++) {
            std::string inputFilename22 = std::string(dirPath) + "/" + *iter;
            std::string outputFilename22 = std::string(outputFilename) + "/" + *iter + ".npy";

            FILE *output_file_test;
            if ((output_file_test = fopen(outputFilename22.c_str(), "rb")) != NULL) {
                std::cerr<<"npy file exists <" << outputFilename22 << ">" << " skip it" <<std::endl;
                fclose(output_file_test);
                continue;
            }

            int ret = resize(inputFilename22.c_str(), outputFilename22, xsize, ysize);
            if (ret != 0)
                continue;

            count++;
        }
        ftime(&t);
        long t2 = t.time * 1000 + t.millitm;

        if (count != 0) {
            std::cout << "process iamges " << count << " for time " << t2 - t1 << " millis at "<< t2 << std::endl;
        }
        usleep(5000);
    }
    return 0;
}
