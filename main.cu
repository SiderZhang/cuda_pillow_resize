#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <stdio.h>
#include <vector>
#include <string.h>
#include "npy.h"

#include "libcuda.h"

#include <sys/timeb.h>
#include <unistd.h>
#include <stdlib.h>
#include <sys/stat.h>

#define PRECISION_BITS (32 - 8 - 2)


typedef signed char         INT8, *PINT8;
typedef signed short        INT16, *PINT16;
typedef signed int          INT32, *PINT32;
typedef unsigned char       UINT8, *PUINT8;
typedef unsigned short      UINT16, *PUINT16;
typedef unsigned int        UINT32, *PUINT32;

#define LANZOS_INTERPOLATION 1
#define BILINEAR_INTERPOLATION 2
#define BICUBIC_INTERPOLATION 3
#define BOX_INTERPOLATION 4
#define HAMMING_INTERPOLATION 5

struct filter {
    double (*filter)(double x);
    double support;
};

__device__ __forceinline__ double
box_filter(double x) {
    if (x > -0.5 && x <= 0.5) {
        return 1.0;
    }
    return 0.0;
}

__device__ double
bilinear_filter(double x) {
    if (x < 0.0) {
        x = -x;
    }
    if (x < 1.0) {
        return 1.0 - x;
    }
    return 0.0;
}

__device__ double
hamming_filter(double x) {
    if (x < 0.0) {
        x = -x;
    }
    if (x == 0.0) {
        return 1.0;
    }
    if (x >= 1.0) {
        return 0.0;
    }
    x = x * M_PI;
    return sin(x) / x * (0.54f + 0.46f * cos(x));
}

__device__ double
bicubic_filter(double x) {
    /* https://en.wikipedia.org/wiki/Bicubic_interpolation#Bicubic_convolution_algorithm
     */
#define a -0.5
    if (x < 0.0) {
        x = -x;
    }
    if (x < 1.0) {
        return ((a + 2.0) * x - (a + 3.0)) * x * x + 1;
    }
    if (x < 2.0) {
        return (((x - 5) * x + 8) * x - 4) * a;
    }
    return 0.0;
#undef a
}

__device__ __forceinline__ double
sinc_filter(double x) {
    if (x == 0.0) {
        return 1.0;
    }
    x = x * M_PI;
    return sin(x) / x;
}

__device__ __forceinline__ double
lanczos_filter(double x) {
    /* truncated sinc */
    if (-3.0 <= x && x < 3.0) {
        return sinc_filter(x) * sinc_filter(x / 3);
    }
    return 0.0;
}


__device__ static struct filter BOX_D = {box_filter, 0.5};
__device__ static struct filter BILINEAR_D = {bilinear_filter, 1.0};
__device__ static struct filter HAMMING_D = {hamming_filter, 1.0};
__device__ static struct filter BICUBIC_D = {bicubic_filter, 2.0};
__device__ static struct filter LANCZOS_D = {lanczos_filter, 3.0};

static struct filter BOX_H = {box_filter, 0.5};
static struct filter BILINEAR_H = {bilinear_filter, 1.0};
static struct filter HAMMING_H = {hamming_filter, 1.0};
static struct filter BICUBIC_H = {bicubic_filter, 2.0};
static struct filter LANCZOS_H = {lanczos_filter, 3.0};

__global__ void test(int step, int* boundsp, int size) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= size) {
        return;
    }

    printf("step: %d, id: %d, bound: %d\n", step, id, boundsp[id]);
}


__global__ void coeffs(int inSize, int in0, int in1,
                       int outSize, int *boundsp, double *kkp, int interpolation_mode) {
    double scale, filterscale;
    double center, ww, ss;
    int x, ksize, xmin, xmax;
    int *bounds;
    double *kk, *k;


    /* prepare for horizontal stretch */
    filterscale = scale = (double)(in1 - in0) / outSize;
    if (filterscale < 1.0) {
        filterscale = 1.0;
    }

    /* determine support size (length of resampling filter) */
    struct filter *filterp = NULL;

    switch (interpolation_mode){
        case 1:
            filterp = &LANCZOS_D;
            break;
        case 2:
            filterp = &BILINEAR_D;
            break;
        case 3:
            filterp = &BICUBIC_D;
            break;
        case 4:
            filterp = &BOX_D;
            break;
        case 5:
            filterp = &HAMMING_D;
            break;
        default:
            printf("unsupported interpolation mode %d", interpolation_mode);
            return;
    }

    double support = filterp->support * filterscale;
//    double support = supportFacter * filterscale;

    /* maximum number of coeffs */
    ksize = (int)ceil(support) * 2 + 1;

    // check for overflow
    if (outSize > INT_MAX / (ksize * (int)sizeof(double))) {
        return;
    }

    /* coefficient buffer */
    /* malloc check ok, overflow checked above */
    kk = kkp;

    /* malloc check ok, ksize*sizeof(double) > 2*sizeof(int) */
    bounds = boundsp;
    int _xx = blockIdx.x * blockDim.x + threadIdx.x;
    if (_xx >= outSize) {
        return;
    }

    for (unsigned int xx = _xx;xx < outSize;xx += gridDim.x * blockDim.x) {
        center = in0 + (xx + 0.5) * scale;
        ww = 0.0;
        ss = 1.0 / filterscale;
        // Round the value
        xmin = (int) (center - support + 0.5);
        if (xmin < 0) {
            xmin = 0;
        }
        // Round the value
        xmax = (int) (center + support + 0.5);
        if (xmax > inSize) {
            xmax = inSize;
        }
        xmax -= xmin;
        k = &kk[xx * ksize];
        for (x = 0; x < xmax; x++) {

//            double filteringX = (x + xmin - center + 0.5) * ss;
//
//            double w = 0;//filterp->filter((x + xmin - center + 0.5) * ss);
//
//            if (filteringX < 0.0) {
//                filteringX = -filteringX;
//            }
//            if (filteringX < 1.0) {
//                w = 1.0 - filteringX;
//            } else {
//                w = 0.0;
//            }
            double w = filterp->filter((x + xmin - center + 0.5) * ss);
//            double w = pFilterFunc((x + xmin - center + 0.5) * ss);

            k[x] = w;
            ww += w;
        }
        for (x = 0; x < xmax; x++) {
            if (ww != 0.0) {
                k[x] /= ww;
            }
        }
        // Remaining values should stay empty if they are used despite of xmax.
        for (; x < ksize; x++) {
            k[x] = 0;
        }

        bounds[xx * 2 + 0] = xmin;
        bounds[xx * 2 + 1] = xmax;
    }
}


__global__ void normalize_coeffs(unsigned int outSize, int ksize, double *prekk) {
    INT32 *kk;


    // use the same buffer for normalized coefficients
    kk = (INT32 *)prekk;

    unsigned int _x = blockIdx.x * blockDim.x + threadIdx.x;
    if (_x >= outSize * ksize) {
        return;
    }


    for (unsigned int x = _x;x < outSize * ksize;x += gridDim.x * blockDim.x) {
        if (prekk[x] < 0) {
            kk[x] = (int) (-0.5 + prekk[x] * (1 << PRECISION_BITS));
        } else {
            kk[x] = (int) (0.5 + prekk[x] * (1 << PRECISION_BITS));
        }
    }
}

__global__ void shift_ysize(int *boundsp, int ysize) {
    unsigned int _i = blockIdx.x * blockDim.x + threadIdx.x;
    if (_i >= ysize) {
        return;
    }
//    printf("Bound %d %d\n", boundsp[i * 2], boundsp[0]);

    // Shift bounds for vertical pass
    for (unsigned int i = _i;i < ysize;i += gridDim.x * blockDim.x) {
        boundsp[i * 2] -= boundsp[0];
    }
}

__global__ void build_result_horiz(unsigned int srcXsize, unsigned int channelCount, unsigned int xsize, unsigned int ysize,
                                   unsigned char* input, unsigned char* output, int ksize, int *bounds, double *prekk, unsigned char* _lookups) {
    unsigned int _id = blockIdx.x * blockDim.x + threadIdx.x;
    if (_id >= xsize * ysize) {
        return;
    }

    for (unsigned int id = _id;id < xsize * ysize;id += gridDim.x * blockDim.x) {

        unsigned int xx = id % xsize;
        unsigned int yy = id / xsize;

        int ss0, ss1, ss2;
        INT32 *kk = (INT32 *) prekk;

        int xmin = bounds[xx * 2 + 0];
        int xmax = bounds[xx * 2 + 1];
        INT32 *k = &kk[xx * ksize];
        ss0 = ss1 = ss2 = 1 << (PRECISION_BITS - 1);

        for (int x = 0; x < xmax; x++) {
//            int s = yy * srcXsize * channelCount + (x + xmin) * channelCount;
//            if (s < 0 || s >= srcXsize * ysize * channelCount) {
//                printf("invalid index: %d, %d, %d, %d, %d, %d\n", yy, channelCount, x, xmax, xmin, s);
//            }

            ss0 += ((UINT8) input[yy * srcXsize * channelCount + (x + xmin) * channelCount + 0]) *
                   k[x];
            ss1 += ((UINT8) input[yy * srcXsize * channelCount + (x + xmin) * channelCount + 1]) *
                   k[x];
            ss2 += ((UINT8) input[yy * srcXsize * channelCount + (x + xmin) * channelCount + 2]) *
                   k[x];
        }

        UINT8 *lookups = &_lookups[640];
        UINT8 ss0_1 = lookups[ss0 >> PRECISION_BITS];
        UINT8 ss1_1 = lookups[ss1 >> PRECISION_BITS];
        UINT8 ss2_1 = lookups[ss2 >> PRECISION_BITS];

        output[yy * xsize * channelCount + xx * channelCount + 0] = ss0_1;
        output[yy * xsize * channelCount + xx * channelCount + 1] = ss1_1;
        output[yy * xsize * channelCount + xx * channelCount + 2] = ss2_1;
    }
}

__global__ void build_result_vert(unsigned int srcXsize, unsigned int channelCount, unsigned int xsize, unsigned int ysize,
                                  unsigned char* input, unsigned char* output, int ksize, int *bounds, double *prekk, unsigned char* _lookups) {
    unsigned int _id = blockIdx.x * blockDim.x + threadIdx.x;
    if (_id >= xsize * ysize) {
        return;
    }

    for (unsigned int id = _id;id < xsize * ysize;id += gridDim.x * blockDim.x) {
        unsigned int xx = id % xsize;
        unsigned int yy = id / xsize;

        int ss0, ss1, ss2;
        INT32 *kk = (INT32 *) prekk;

        INT32 *k = &kk[yy * ksize];
        int ymin = bounds[yy * 2 + 0];
        int ymax = bounds[yy * 2 + 1];

        ss0 = ss1 = ss2 = 1 << (PRECISION_BITS - 1);
        for (int y = 0; y < ymax; y++) {
            ss0 += ((UINT8) input[(y + ymin) * srcXsize * channelCount + xx * channelCount + 0]) * k[y];
            ss1 += ((UINT8) input[(y + ymin) * srcXsize * channelCount + xx * channelCount + 1]) * k[y];
            ss2 += ((UINT8) input[(y + ymin) * srcXsize * channelCount + xx * channelCount + 2]) * k[y];
        }

        UINT8 *lookups = &_lookups[640];
        UINT8 ss0_1 = lookups[ss0 >> PRECISION_BITS];
        UINT8 ss1_1 = lookups[ss1 >> PRECISION_BITS];
        UINT8 ss2_1 = lookups[ss2 >> PRECISION_BITS];
        output[yy * xsize * channelCount + xx * channelCount + 0] = ss0_1;
        output[yy * xsize * channelCount + xx * channelCount + 1] = ss1_1;
        output[yy * xsize * channelCount + xx * channelCount + 2] = ss2_1;
    }
}

float* corp(float* src, unsigned int input_size, unsigned int corp_size, unsigned int channels){
    float* target;
    hipMalloc(&target, corp_size * corp_size * channels * sizeof(float));
    unsigned int left = (input_size - corp_size) / 2;
    unsigned int top = (input_size - corp_size) / 2;
    for (int i = 0;i < corp_size;i++) {
        unsigned int src_start = left + input_size * (i + top);
        unsigned int dest_start = corp_size * i;
        hipMemcpy(&target[dest_start + corp_size * corp_size * 0], &src[src_start + input_size * input_size * 0], corp_size * sizeof(float), hipMemcpyDeviceToDevice);
        hipMemcpy(&target[dest_start + corp_size * corp_size * 1], &src[src_start + input_size * input_size * 1], corp_size * sizeof(float), hipMemcpyDeviceToDevice);
        hipMemcpy(&target[dest_start + corp_size * corp_size * 2], &src[src_start + input_size * input_size * 2], corp_size * sizeof(float), hipMemcpyDeviceToDevice);
    }
    hipFree(src);
    return target;
}

__global__ void rescale_normalize_d(unsigned char *data, float* result, unsigned int xsize, unsigned int ysize,
                                    float mean0, float mean1, float mean2, float std0, float std1, float std2) {
    unsigned int _id = blockIdx.x * blockDim.x + threadIdx.x;


    if (_id >= xsize * ysize) {
        return;
    }

    for (unsigned int id = _id;id < xsize * ysize;id += gridDim.x * blockDim.x) {
        // rescale
        double pixel0 = ((double)data[id * 3 + 0]) * 1.0 / 255;
        double pixel1 = ((double)data[id * 3 + 1]) * 1.0 / 255;
        double pixel2 = ((double)data[id * 3 + 2]) * 1.0 / 255;

        // normalize
        pixel0 = (pixel0 - mean0) / std0;
        pixel1 = (pixel1 - mean1) / std1;
        pixel2 = (pixel2 - mean2) / std2;

        // transpose
        result[id + 0 * xsize * ysize] = (float)pixel0;
        result[id + 1 * xsize * ysize] = (float)pixel1;
        result[id + 2 * xsize * ysize] = (float)pixel2;
    }
}

UINT8 lookups_h[1280] = {
        0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
        0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
        0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
        0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
        0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
        0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
        0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
        0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
        0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
        0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
        0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
        0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
        0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
        0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
        0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
        0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
        0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
        0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
        0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
        0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
        0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
        0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
        0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
        0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
        0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
        0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
        0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
        0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
        0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
        0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
        0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
        0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
        0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
        0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
        0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
        0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
        0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
        0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   1,   2,   3,   4,   5,
        6,   7,   8,   9,   10,  11,  12,  13,  14,  15,  16,  17,  18,  19,  20,  21,  22,
        23,  24,  25,  26,  27,  28,  29,  30,  31,  32,  33,  34,  35,  36,  37,  38,  39,
        40,  41,  42,  43,  44,  45,  46,  47,  48,  49,  50,  51,  52,  53,  54,  55,  56,
        57,  58,  59,  60,  61,  62,  63,  64,  65,  66,  67,  68,  69,  70,  71,  72,  73,
        74,  75,  76,  77,  78,  79,  80,  81,  82,  83,  84,  85,  86,  87,  88,  89,  90,
        91,  92,  93,  94,  95,  96,  97,  98,  99,  100, 101, 102, 103, 104, 105, 106, 107,
        108, 109, 110, 111, 112, 113, 114, 115, 116, 117, 118, 119, 120, 121, 122, 123, 124,
        125, 126, 127, 128, 129, 130, 131, 132, 133, 134, 135, 136, 137, 138, 139, 140, 141,
        142, 143, 144, 145, 146, 147, 148, 149, 150, 151, 152, 153, 154, 155, 156, 157, 158,
        159, 160, 161, 162, 163, 164, 165, 166, 167, 168, 169, 170, 171, 172, 173, 174, 175,
        176, 177, 178, 179, 180, 181, 182, 183, 184, 185, 186, 187, 188, 189, 190, 191, 192,
        193, 194, 195, 196, 197, 198, 199, 200, 201, 202, 203, 204, 205, 206, 207, 208, 209,
        210, 211, 212, 213, 214, 215, 216, 217, 218, 219, 220, 221, 222, 223, 224, 225, 226,
        227, 228, 229, 230, 231, 232, 233, 234, 235, 236, 237, 238, 239, 240, 241, 242, 243,
        244, 245, 246, 247, 248, 249, 250, 251, 252, 253, 254, 255, 255, 255, 255, 255, 255,
        255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
        255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
        255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
        255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
        255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
        255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
        255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
        255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
        255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
        255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
        255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
        255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
        255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
        255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
        255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
        255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
        255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
        255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
        255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
        255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
        255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
        255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
        255, 255, 255, 255, 255,
};
float* image_preprocess(unsigned char* input, unsigned int source_xsize, unsigned int source_ysize, unsigned int target_xsize, unsigned int target_ysize, unsigned int channels, int interpolation_mode, float* mean, float* std) {
    int *bounds_horiz_d;
    double *kk_horiz_d;
    int ksize_horiz;

    int  *bounds_vert_d;
    double *kk_vert_d;
    int ksize_vert;

    float box[4] = {0, 0, 1.0f * source_xsize, 1.0f * source_ysize};

    double support;
    struct filter *filterp;

    switch (interpolation_mode){
        case 1:
            filterp = &LANCZOS_H;
            break;
        case 2:
            filterp = &BILINEAR_H;
            break;
        case 3:
            filterp = &BICUBIC_H;
            break;
        case 4:
            filterp = &BOX_H;
            break;
        case 5:
            filterp = &HAMMING_H;
            break;
        default:
            printf("unsupported interpolation mode %d", interpolation_mode);
            return NULL;
    }

    double filterscale_horiz = (double)(box[2] - box[0]) / target_xsize;
    if (filterscale_horiz < 1.0) {
        filterscale_horiz = 1.0;
    }
    ksize_horiz = (int)ceil(filterscale_horiz * filterp->support) * 2 + 1;

    hipMalloc(&kk_horiz_d, target_xsize * ksize_horiz * sizeof(double));
    hipMalloc(&bounds_horiz_d, target_xsize * 2 * sizeof(int));

    coeffs<<<256, 256>>>(source_xsize,
                         box[0],
                         box[2],
                         target_xsize,
                         bounds_horiz_d,
                         kk_horiz_d,
                         interpolation_mode);

    hipError_t error =  hipGetLastError();
    if (error != hipError_t::hipSuccess) {
        std::cerr<<hipGetErrorString(error)<<std::endl;
        return NULL;
    }

    normalize_coeffs<<<target_xsize, ksize_horiz>>>(target_xsize, ksize_horiz, kk_horiz_d);

    error =  hipGetLastError();
    if (error != hipError_t::hipSuccess) {
        std::cerr<<hipGetErrorString(error)<<std::endl;
        return NULL;
    }

    double filterscale_vert = (double)(box[3] - box[1]) / target_ysize;
    if (filterscale_vert < 1.0) {
        filterscale_vert = 1.0;
    }
    ksize_vert = (int)ceil(filterscale_vert * filterp->support) * 2 + 1;
    hipMalloc(&kk_vert_d, target_ysize * ksize_vert * sizeof(double));
    hipMalloc(&bounds_vert_d, target_ysize * 2 * sizeof(int));

    error = hipGetLastError();
    if (error != hipError_t::hipSuccess) {
        std::cerr<<hipGetErrorString(error)<<std::endl;
        return NULL;
    }

    coeffs<<<256, 256>>>(source_ysize,
                         box[1],
                         box[3],
                         target_ysize,
                         bounds_vert_d,
                         kk_vert_d,
                         interpolation_mode);

    error = hipGetLastError();
    if (error != hipError_t::hipSuccess) {
        std::cerr<<hipGetErrorString(error)<<std::endl;
        return NULL;
    }

    normalize_coeffs<<<target_ysize, ksize_vert>>>(target_ysize, ksize_vert, kk_vert_d);

    error = hipGetLastError();
    if (error != hipError_t::hipSuccess) {
        std::cerr<<hipGetErrorString(error)<<std::endl;
        return NULL;
    }

    shift_ysize<<<256, 256>>>(bounds_vert_d, target_ysize);

    error = hipGetLastError();
    if (error != hipError_t::hipSuccess) {
        std::cerr<<hipGetErrorString(error)<<std::endl;
        return NULL;
    }

    unsigned char* temp;
    hipMalloc(&temp, target_xsize * source_ysize * channels * sizeof(unsigned char));

    unsigned char* result_pixel_data_d;
    hipMalloc(&result_pixel_data_d, target_xsize * target_ysize * channels * sizeof(unsigned char));

    unsigned char* looksup_d;
    hipMalloc(&looksup_d, 1280 * sizeof(unsigned char));
    hipMemcpy(looksup_d, lookups_h, 1280 * sizeof(unsigned char), hipMemcpyHostToDevice);

    build_result_horiz<<<256, 1024>>>(source_xsize, channels, target_xsize, source_ysize, input, temp, ksize_horiz, bounds_horiz_d, kk_horiz_d, looksup_d);

    error = hipGetLastError();
    if (error != hipError_t::hipSuccess) {
        std::cerr<<hipGetErrorString(error)<<std::endl;
        return NULL;
    }

    build_result_vert<<<256, 1024>>>(target_xsize, channels, target_xsize, target_ysize, temp, result_pixel_data_d, ksize_vert, bounds_vert_d, kk_vert_d, looksup_d);

    error = hipGetLastError();
    if (error != hipError_t::hipSuccess) {
        std::cerr<<hipGetErrorString(error)<<std::endl;
        return NULL;
    }

    float* normalized_pixel_data_d;
    hipMalloc(&normalized_pixel_data_d, target_xsize * target_ysize * channels * sizeof(float));

    rescale_normalize_d<<<256, 1024>>>(result_pixel_data_d, normalized_pixel_data_d, target_xsize, target_ysize,
                                       mean[0], mean[1], mean[2], std[0], std[1], std[2]);

    error = hipGetLastError();
    if (error != hipError_t::hipSuccess) {
        std::cerr<<hipGetErrorString(error)<<std::endl;
        return NULL;
    }

    hipFree(temp);
    hipFree(looksup_d);
    hipFree(result_pixel_data_d);

    hipFree(bounds_horiz_d);
    hipFree(kk_horiz_d);
    hipFree(bounds_vert_d);
    hipFree(kk_vert_d);

    error = hipGetLastError();
    if (error != hipError_t::hipSuccess) {
        std::cerr<<hipGetErrorString(error)<<std::endl;
        return NULL;
    }

    return normalized_pixel_data_d;
}

float* vit_preprocess(unsigned char* input, unsigned int source_xsize, unsigned int source_ysize, unsigned int vit_size, unsigned int channels) {
    float mean[3] = {0.5, 0.5, 0.5};
    float std[3] = {0.5, 0.5, 0.5};
    int interpolation_mode = BILINEAR_INTERPOLATION;
    return image_preprocess(input, source_xsize, source_ysize, vit_size, vit_size, channels, interpolation_mode, mean, std);
}

float* deit_preprocess(unsigned char* input, unsigned int source_xsize, unsigned int source_ysize, unsigned int vit_size, unsigned int corp_size, unsigned int channels) {
    float mean[3] = {0.485, 0.456, 0.406};
    float std[3] = {0.229, 0.224, 0.225};
    int interpolation_mode = BICUBIC_INTERPOLATION;
    float* resizedImage = image_preprocess(input, source_xsize, source_ysize, vit_size, vit_size, channels, interpolation_mode, mean, std);

    return resizedImage;
//    return corp(resizedImage, vit_size, corp_size, channels);
}

float* vis_preprocess(unsigned char* input, unsigned int source_xsize, unsigned int source_ysize, unsigned int vit_size, unsigned int channels) {
    float mean[3] = {0.48145466, 0.4578275, 0.40821073};
    float std[3] = {0.26862954, 0.26130258, 0.27577711};
    int interpolation_mode = BICUBIC_INTERPOLATION;
    float* resizedImage = image_preprocess(input, source_xsize, source_ysize, vit_size, vit_size, channels, interpolation_mode, mean, std);
    return resizedImage;
}

void onImageRead(unsigned char* input, unsigned char **output_buffer, unsigned int width, unsigned int height, unsigned int channels){
    hipMalloc(output_buffer, width * height * channels * sizeof(unsigned char));
    hipMemcpy(*output_buffer, input, width * height * channels * sizeof(unsigned char), hipMemcpyHostToDevice);
}

void to_npy(const char* filename, std::string& outputFilename, float* array, unsigned int size, unsigned int channels) {
    const std::vector<unsigned long int> leshape11{3, size, size};
    std::vector<float> deit_vec(array, array + size * size * channels);
//    std::vector<float> deit_vec;
    npy::npy_data<float> data11;
    data11.data = deit_vec;
    data11.shape = leshape11;
    data11.fortran_order = false;
    std::string tmpFileName = outputFilename + ".tmp";
    write_npy(tmpFileName, data11);
    rename(tmpFileName.c_str(), outputFilename.c_str());
}


int image_process(const char* filename, std::string& output_filename_prefix, unsigned int vit_size, unsigned int vis_size, unsigned int deit_size, unsigned int corp_size) {
    unsigned int source_xsize;
    unsigned int source_ysize;
    unsigned int channels;

    unsigned char* input;
    int ret = load_image_file(filename, &input, source_xsize, source_ysize, channels);
    if (ret == -1) {
        return -1;
    }

    float* deit_pixel_data_d = deit_preprocess(input, source_xsize, source_ysize, deit_size, corp_size, channels);
    float* deit_result_h;
    deit_result_h = (float*)malloc(corp_size * corp_size * channels * sizeof(float));
    hipMemcpy(deit_result_h, deit_pixel_data_d, corp_size * corp_size * channels * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(deit_pixel_data_d);
    std::string deit_output_filename = output_filename_prefix + "_deit.npy";
    to_npy(filename, deit_output_filename, deit_result_h, corp_size, channels);
    free(deit_result_h);

    float* vis_pixel_data_d = vis_preprocess(input, source_xsize, source_ysize, vis_size, channels);
    float* vis_reuslt_h;
    vis_reuslt_h = (float*)malloc(vis_size * vis_size * channels * sizeof(float));
    hipMemcpy(vis_reuslt_h, vis_pixel_data_d, vis_size * vis_size * channels * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(vis_pixel_data_d);
    std::string vis_output_filename = output_filename_prefix + "_vis.npy";
    to_npy(filename, vis_output_filename, vis_reuslt_h, vis_size, channels);
    free(vis_reuslt_h);

    float* vit_pixel_data_d = vit_preprocess(input, source_xsize, source_ysize, vit_size, channels);
    float* vit_reuslt_h;
    vit_reuslt_h = (float*)malloc(vit_size * vit_size * channels * sizeof(float));
    hipMemcpy(vit_reuslt_h, vit_pixel_data_d, vit_size * vit_size * channels * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(vit_pixel_data_d);
    std::string vit_output_filename = output_filename_prefix + "_vit.npy";
    to_npy(filename, vit_output_filename, vit_reuslt_h, vit_size, channels);
    free(vit_reuslt_h);

    hipFree(input);
    hipDeviceSynchronize();

    return 0;
}

int main(int argc, char *argv[]) {
    if (argc < 2) {
        std::cerr<<"arguments: input_dir output_dir"<<std::endl;
        return -1;
    }
    char* input_dir = argv[1];
    char* output_dir = argv[2];

    while (true) {
        std::vector<std::string> fileNames;
        readDir(input_dir, fileNames, std::string(output_dir));

        timeb t;
        ftime(&t);
        long t1 = t.time * 1000 + t.millitm;
        int count = 0;
        for (auto iter = fileNames.begin();iter != fileNames.end();iter++) {
            std::string input_file_abs_path = std::string(input_dir) + "/" + *iter;
            std::string output_file_abs_path = std::string(output_dir) + "/" + *iter;

            int ret = image_process(input_file_abs_path.c_str(), output_file_abs_path, 224, 224, 256, 256);
            if (ret != 0)
                continue;

            ret = remove(input_file_abs_path.c_str());
            if (ret != 0) {
                std::cerr<<"failed to delete processed image file "<<input_file_abs_path.c_str()<<std::endl;
                continue;
            }

            count++;
        }
        ftime(&t);
        long t2 = t.time * 1000 + t.millitm;

        if (count != 0) {
            std::cout << "process images " << count << " for time " << t2 - t1 << " millis at "<< t2 << std::endl;
        }
        usleep(10000);
    }

//    unsigned char* data;
//    read2("/home/siderzhang/file/9.jpg", &data);

//    if (argc < 1) {
//        std::cerr<<"arguments: input_filename"<<std::endl;
//        return -1;
//    }
//    const char* input_file = argv[1];
//    std::string output_filename_prefix = std::string("hello");
//    image_process(input_file, output_filename_prefix, 224, 224, 256, 256);
//    return 0;
}
